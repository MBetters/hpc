
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_kernel(){
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	printf("Hello from block %d, thread %d of the GPU!\n", bid, tid);
}

extern "C" void hello(){
	// do stuff here
	printf("Executing kernel...\n");
	hello_kernel<<<2,2>>>();
	hipDeviceSynchronize();
}


